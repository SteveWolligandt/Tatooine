#include "hip/hip_runtime.h"
#include <tatooine/cuda/buffer.cuh>
#include <tatooine/cuda/tex.cuh>
#include <tatooine/cuda/coordinate_conversion.cuh>

#include <catch2/catch.hpp>
#include <fstream>
#include <iostream>

#include "write_ppm.h"

//==============================================================================
namespace tatooine {
namespace cuda {
namespace test {
//==============================================================================
auto make_test_texture(const size_t width, const size_t height) {
  std::vector<float> h_original(width * height * 4, 0.0f);
  for (size_t y = 0; y < height / 2; ++y) {
    for (size_t x = 0; x < width / 2; ++x) {
      size_t i          = x + width * y;
      h_original[i * 4] = 1;
    }
  }
  for (size_t y = 0; y < height / 2; ++y) {
    for (size_t x = width / 2; x < width; ++x) {
      size_t i              = x + width * y;
      h_original[i * 4 + 1] = 1;
    }
  }
  for (size_t y = height / 2; y < height; ++y) {
    for (size_t x = 0; x < width / 2; ++x) {
      size_t i              = x + width * y;
      h_original[i * 4 + 2] = 1;
    }
  }
  for (size_t y = height / 2; y < height; ++y) {
    for (size_t x = width / 2; x < width; ++x) {
      size_t i              = x + width * y;
      h_original[i * 4 + 1] = 1;
      h_original[i * 4 + 2] = 1;
    }
  }
  return h_original;
}

__global__ void kernel0(tex<float,4,2> t, buffer<float> out,
                        float theta) {
  const auto   globalIdx = make_uint2(blockIdx.x * blockDim.x + threadIdx.x,
                                    blockIdx.y * blockDim.y + threadIdx.y);
  const auto texres = t.resolution();
  const size_t plainIdx  = globalIdx.y * texres.x + globalIdx.x;
  if (globalIdx.x >= texres.x || globalIdx.y >= texres.y) { return; }
  // calculate normalized texture coordinates
  const auto uv = global_idx_to_uv2(globalIdx, texres);

  // transform texture coordinates
  const auto tuv = make_float2(
      (uv.x - 0.5) * cosf(theta) - (uv.y - 0.5) * sinf(theta) + 0.5f,
      (uv.y - 0.5) * cosf(theta) + (uv.x - 0.5) * sinf(theta) + 0.5f);

  // sample texture and assign to output array
  const auto col        = t(tuv.x, tuv.y);
  out[plainIdx * 3]     = col.x;
  out[plainIdx * 3 + 1] = col.y;
  out[plainIdx * 3 + 2] = col.z;
}
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
TEST_CASE("cuda_tex0", "[cuda][tex]") {
  const size_t width = 1024, height = 1024;
  const auto   h_original =
      make_test_texture(width, height);  // creates float-rgba texture with 4
                                         // differently colored areas
  write_ppm("untransformed.ppm", h_original, width, height, 4);

  // upload texture data to hipArray
  tex<float, 4, 2> d_tex{h_original, width, height};

  // create device memory for output of transformed texture
  buffer<float> d_out(width * height * 3);

  // call kernel
  const dim3 numthreads(256, 256);
  const dim3 numblocks(width / numthreads.x + 1, height / numthreads.y + 1);
  kernel0<<<numthreads, numblocks>>>(d_tex, d_out, M_PI / 4);

  // download transformed texture data and write
  write_ppm("transformed.ppm", d_out.download(), width, height, 3);
}

//==============================================================================
__global__ void kernel1(tex<float,4,2> t, buffer<float> out) {
  const auto   globalIdx = make_uint2(blockIdx.x * blockDim.x + threadIdx.x,
                                    blockIdx.y * blockDim.y + threadIdx.y);
  const auto texres = t.resolution();
  const size_t plainIdx  = globalIdx.y * texres.x + globalIdx.x;
  if (globalIdx.x >= texres.x || globalIdx.y >= texres.y) { return; }
  // calculate normalized texture coordinates
  const auto uv = global_idx_to_uv2(globalIdx, texres);

  // sample texture and assign to output array
  const auto col        = t(uv.x, uv.y);
  out[plainIdx * 3]     = col.x;
  out[plainIdx * 3 + 1] = col.y;
  out[plainIdx * 3 + 2] = col.z;
}
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
TEST_CASE("cuda_tex1", "[cuda][tex]") {
  const size_t width = 1024, height = 1024;
  const auto   h_tex =
      make_test_texture(width, height);  // creates float-rgba texture with 4
                                         // differently colored areas

  // upload texture data
  tex<float, 4, 2> d_tex{h_tex, true, linear, border, width, height};

  // create device memory for output of transformed texture
  buffer<float> d_out(width * height * 3);

  // call kernel
  const dim3 numthreads(256, 256);
  const dim3 numblocks(width / numthreads.x + 1, height / numthreads.y + 1);
  kernel1<<<numthreads, numblocks>>>(d_tex, d_out);
  auto h_out = d_out.download();
  for (size_t i = 0; i < width * height; ++i) {
    for (size_t j = 0; j < 3; ++j) {
      REQUIRE(h_tex[i * 4 + j] == h_out[i * 3 + j]);
    }
  }
}

//==============================================================================
}  // namespace test
}  // namespace cuda
}  // namespace tatooine
//==============================================================================
