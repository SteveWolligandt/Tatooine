#include "hip/hip_runtime.h"
#include <tatooine/cuda/global_buffer.h>
#include <tatooine/cuda/texture_buffer.h>
#include <catch2/catch.hpp>
#include <iostream>

//==============================================================================
namespace tatooine {
namespace gpu {
namespace test {
//==============================================================================

__global__ void test_kernel1(hipTextureObject_t tex, float *out, size_t  width,
                             size_t  height) {
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    // calculate normalized texture coordinates
    const float u      = x / float(width - 1);
    const float v      = y / float(height - 1);
    out[y * width + x] = tex2D<float>(tex, x,y);
  }
}
//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
TEST_CASE("cuda_texture_buffer_upload_download_vector",
          "[cuda][texture_buffer][upload][download][vector]") {
  const size_t       width = 10, height = 10;
  std::vector<float> h_tex(width * height);
  for (size_t i = 0; i < width * height; ++i) { h_tex[i] = i; }
  cuda::texture_buffer<float, 1, 2> d_tex(h_tex, width, height);
  cuda::global_buffer<float>        d_out(width * height);

  const dim3 dimBlock(16, 16);
  const dim3 dimGrid(width / dimBlock.x + 1, height / dimBlock.y + 1);
  test_kernel1<<<dimBlock, dimGrid>>>(d_tex.device_ptr(), d_out.device_ptr(),
                                      width, height);
  hipDeviceSynchronize();

  const auto h_out = d_out.download();
  for (size_t i = 0; i < h_tex.size(); ++i) {
    INFO("i = " << i);
    REQUIRE(h_out[i] == h_tex[i]);
  }
}
//==============================================================================
__global__ void test_kernel2(hipTextureObject_t tex, float *out, size_t width,
                             size_t height) {
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    // calculate normalized texture coordinates
    const float  u          = x / float(width - 1);
    const float  v          = y / float(height - 1);
    const auto   sample     = tex2D<float2>(tex, u, v);

    const size_t global_idx = x + y * width;
    out[global_idx * 2]     = sample.x;
    out[global_idx * 2 + 1] = sample.y;
  }
}
//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
TEST_CASE("cuda_texture_buffer_upload_download_vector2",
          "[cuda][texture_buffer][upload][download][vector]") {
  const size_t       width = 10, height = 10;
  const float        val = 5.0f;
  std::vector<float> h_tex(width * height * 2, val);
  for (size_t i = 0; i < width * height; ++i) { h_tex[i * 2 + 1] = i; }
  cuda::texture_buffer<float, 2, 2> d_tex(h_tex, width, height);
  cuda::global_buffer<float>        d_out(width * height * 2);

  const dim3 dimBlock(16, 16);
  const dim3 dimGrid(width / dimBlock.x + 1, height / dimBlock.y + 1);
  test_kernel2<<<dimBlock, dimGrid>>>(d_tex.device_ptr(), d_out.device_ptr(),
                                      width, height);
  hipDeviceSynchronize();

  const auto h_out = d_out.download();
  for (size_t i = 0; i < h_tex.size(); ++i) {
    INFO("i = " << i);
    REQUIRE(h_out[i] == h_tex[i]);
  }
}

//==============================================================================
}  // namespace test
}  // namespace gpu
}  // namespace tatooine
//==============================================================================
