#include "hip/hip_runtime.h"
#include <tatooine/gpu/add.h>
#include <cassert>

//==============================================================================
namespace tatooine {
namespace gpu {
//==============================================================================

__global__
void global_add(int n, float *x, float *y) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i+=stride) {
    x[i] = x[i] + y[i];
  }
}
// - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
void add(std::vector<float>& x, const std::vector<float>& y) {
  assert(x.size() == y.size());
  float *d_x=nullptr, *d_y=nullptr;
  const size_t size = x.size() * sizeof(float);
  hipMalloc((void**)&d_x, size);
  hipMalloc((void**)&d_y, size);
  hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), size, hipMemcpyHostToDevice);

  const int blockSize = 256;
  const int numBlocks = (x.size() + blockSize - 1) / blockSize;
  global_add<<<numBlocks, blockSize>>>(x.size(), d_x, d_y);
  hipDeviceSynchronize();
  hipMemcpy(&x[0], d_x, size, hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

//==============================================================================
} // namespace gpu
} // namespace tatooine
//==============================================================================
